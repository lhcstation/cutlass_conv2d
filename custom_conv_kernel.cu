#include <iostream>
#include <fstream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/convolution.h"
#include "cutlass/util/tensor_view_io.h"


// Data types for input and output tensors
// and computation between elements
using ElementAccumulator = float;                  // Data type of accumulator
using ElementComputeEpilogue = float;              // Data type of epilogue computation (alpha, beta)
using ElementInputA = cutlass::half_t;             // Data type of elements in input tensor
using ElementInputB = cutlass::half_t;             // Data type of elements in input tensor
using ElementOutput = float;                       // Data type of elements in output tensor

using LayoutInputA = cutlass::layout::TensorNHWC;
using LayoutInputB = cutlass::layout::TensorNHWC;
using LayoutOutput = cutlass::layout::TensorNHWC;

// Whether to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// SM architecture number
using SmArch = cutlass::arch::Sm80;

// Threadblock tile shape
using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 64>;

// Warp tile shape
using WarpShape = cutlass::gemm::GemmShape<64, 64, 64>;

// MMA (Tensor Core instruction, in this case) tile shape
using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;

// How the kernel schedules threadblocks
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

// Number of pipeline stages to use
constexpr int NumStages = 3;

// Which iterator algorithm to use: Analytic or Optimized
static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm = cutlass::conv::IteratorAlgorithm::kOptimized;

// Is the output packed or strided
// Use kStride if using strided output
static cutlass::conv::StrideSupport const OutputStride = cutlass::conv::StrideSupport::kUnity;

// The epilogue part of the kernel
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // Data type of output matrix.
    128 / cutlass::sizeof_bits<ElementOutput>::value,  // The number of elements per vectorized
                                                       // memory access. This becomes the vector width of
                                                       // math instructions in the epilogue too.
    ElementAccumulator,                                // Data type of accumulator
    ElementComputeEpilogue>;                           // Data type for alpha/beta in linear combination

// Kernel properties type
using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
  ElementInputA, LayoutInputA,
  ElementInputB, LayoutInputB,
  ElementOutput, LayoutOutput,
  ElementAccumulator,
  MMAOp,
  SmArch,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOp,
  SwizzleThreadBlock,
  NumStages,
  cutlass::arch::OpMultiplyAdd,
  IteratorAlgorithm,
  OutputStride
>::Kernel;

// Type of the actual kernel
using ImplicitGemm = cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  cutlass::Tensor4DCoord input_size;
  cutlass::Tensor4DCoord filter_size;
  cutlass::Tensor4DCoord padding;
  cutlass::MatrixCoord conv_stride;
  cutlass::MatrixCoord dilation;
  bool reference_check;
  bool measure_performance;
  int iterations;
  bool save_workspace;
  ElementComputeEpilogue alpha;
  ElementComputeEpilogue beta;
  bool benchmark;
  std::string tag;

  Options():
    help(false),
    input_size(1, 32, 32, 32),
    filter_size(32, 3, 3, 32),
    padding(1, 1, 1, 1),
    conv_stride(1, 1),
    dilation(1, 1),
    reference_check(false),
    measure_performance(true),
    iterations(20),
    save_workspace(false),
    alpha(1),
    beta(0),
    benchmark(false) { }

  // Verify that the problem size is compatible with CUTLASS's convolution implementation
  bool valid() {

    //
    // CUTLASS attempts to load 128b vectors of cutlass::half_t (F16) elements. Consequently,
    // all pointers, strides, and tensor extents must be divisible by 8 elements.
    //
    int const kAlignment = 8;

    if ((input_size.c() % kAlignment) ||
      (filter_size.n() % kAlignment)) {

      // misaligned tensors
      return false;
    }

    // Invalid padding
    if ((padding.h() != filter_size.h() / 2) ||
      (padding.w() != filter_size.w() / 2)) {

      return false;
    }

    return true;
  }

  /// Update input and filter sizes
  void update(
    cutlass::Tensor4DCoord input_size,
    cutlass::Tensor4DCoord filter_size) {

    this->input_size = input_size;
    this->filter_size = filter_size;

    padding.n() = filter_size.h() / 2;
    padding.h() = filter_size.h() / 2;
    padding.w() = filter_size.w() / 2;
    padding.c() = filter_size.w() / 2;
  }

  // Parse command-line arguments
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    if (cmd.check_cmd_line_flag("ref-check")) {
      reference_check = true;
    }

    if (cmd.check_cmd_line_flag("perf-check")) {
      measure_performance = true;
    }

    if (cmd.check_cmd_line_flag("save-workspace")) {
      save_workspace = true;
    }

    if (cmd.check_cmd_line_flag("benchmark")) {
      benchmark = true;
    }

    cmd.get_cmd_line_argument("n", input_size.n());
    cmd.get_cmd_line_argument("h", input_size.h());
    cmd.get_cmd_line_argument("w", input_size.w());
    cmd.get_cmd_line_argument("c", input_size.c());

    cmd.get_cmd_line_argument("k", filter_size.n());
    cmd.get_cmd_line_argument("r", filter_size.h());
    cmd.get_cmd_line_argument("s", filter_size.w());
    filter_size.c() = input_size.c();

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);

    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("tag", tag);

    if (filter_size.h() == 3 && filter_size.w() == 3) {
      padding = {1, 1, 1, 1};
    }
    else {
      filter_size.h() = 1;
      filter_size.w() = 1;
      padding = {0, 0, 0, 0};
    }
  }


  /// Computes the output tensor size (NPQK)
  cutlass::Tensor4DCoord output_size() const {
    return cutlass::Tensor4DCoord(
      input_size.n(),
      (input_size.h() + padding.n() + padding.h() - filter_size.h()) / conv_stride.row() + 1,
      (input_size.w() + padding.w() + padding.c() - filter_size.w()) / conv_stride.column() + 1,
      filter_size.n());
  }

  /// Compute performance in Gflop/s
  ///
  /// Gflop/s stands for billions (10^9) of
  /// floating-point operations per second (Gflop/s).
  double gflops(double runtime_s) const {

    // Number of multiply-adds = NPQK * CRS
    int64_t fmas = output_size().product() * int64_t(filter_size.h() * filter_size.w() * filter_size.c());

    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};


// CUDA 内核函数
torch::Tensor custom_conv_forward(
    torch::Tensor input, torch::Tensor filter,
    int N, int H, int W, int C,
    int K, int R, int S,
    int pad_h=1, int pad_w=1,
    int stride_h=1, int stride_w=1,
    int dilation_h=1, int dilation_w=1)
{
    cutlass::Tensor4DCoord input_size(N,H,W,C);
    cutlass::Tensor4DCoord filter_size(K,R,S,C);
    cutlass::Tensor4DCoord padding(pad_h, pad_h, pad_w, pad_w);
    cutlass::MatrixCoord conv_stride(stride_h, stride_w);
    cutlass::MatrixCoord dilation(dilation_h, dilation_w);
    ElementComputeEpilogue alpha(1);
    ElementComputeEpilogue beta(0);

    output_size = cutlass::Tensor4DCoord(
      input_size.n(),
      (input_size.h() + padding.n() + padding.h() - filter_size.h()) / conv_stride.row() + 1,
      (input_size.w() + padding.w() + padding.c() - filter_size.w()) / conv_stride.column() + 1,
      filter_size.n());

    cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(input_size);
    cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(filter_size);
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(output_size);
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(output_size);

    const void* input_data_ptr = input.data_ptr();
    size_t torch_data_size = input.numel() * sizeof(ElementInputA);

    // 获取 CUTLASS HostTensor 的设备指针
    void* tensor_a_data_ptr = tensor_a.device_data();
    size_t tensor_a_data_size =tensor_a.capacity() * sizeof(ElementInputA);

    // 使用 hipMemcpy 将数据从 torch::Tensor 复制到 CUTLASS HostTensor
    hipMemcpy(tensor_a_data_ptr, input_data_ptr, torch_data_size, hipMemcpyDeviceToDevice);

    const void* filter_data_ptr = filter.data_ptr();
    size_t filter_data_size = filter.numel() * sizeof(ElementInputB);

    // 获取 CUTLASS HostTensor 的设备指针
    void* tensor_b_data_ptr = tensor_b.device_data();
    size_t tensor_b_data_size =tensor_b.capacity() * sizeof(ElementInputB);

    // 使用 hipMemcpy 将数据从 torch::Tensor 复制到 CUTLASS HostTensor
    hipMemcpy(tensor_b_data_ptr, filter_data_ptr, flter_data_size, hipMemcpyDeviceToDevice);

    cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

    // Split K dimension into 1 partitions
    int split_k_slices = 1;

    // Construct Conv2dProblemSize with user defined output size
    cutlass::conv::Conv2dProblemSize problem_size(
        input_size,
        filter_size,
        padding,
        conv_stride,
        dilation,
        output_size,
        mode,
        split_k_slices
    );

  // Construct ImplicitGemm::Argument structure with conv2d
  // problem size, data pointers, and epilogue values
  typename ImplicitGemm::Arguments arguments{
    problem_size,
    tensor_a.device_ref(),
    tensor_b.device_ref(),
    tensor_c.device_ref(),
    tensor_d.device_ref(),
    {alpha, beta},
  };

  //
  // Initialize CUTLASS Convolution
  //

  ImplicitGemm implicit_gemm_op;

  size_t workspace_size = implicit_gemm_op.get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(status);

  status = implicit_gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(status);

  //
  // Launch initialized CUTLASS kernel
  //
  status = implicit_gemm_op();

  CUTLASS_CHECK(result.status);

    auto shape = tensor_d.extent();
    int n = shape.n();  // Batch size
    int h = shape.h();  // Height
    int w = shape.w();  // Width
    int c = shape.c();  // Channels

    // 创建一个 PyTorch Tensor（NHWC 格式）
    torch::Tensor output = torch::empty({n, h, w, c}, torch::dtype(torch::kFloat32).device(torch::kCUDA));

    // 获取 CUTLASS HostTensor 的数据指针
    const void* cutlass_data_ptr = tensor_d.device_data();
    size_t cutlass_data_size = tensor_d.capacity() * sizeof(ElementOutput);

    // 获取 PyTorch Tensor 的数据指针
    void* torch_data_ptr = output.data_ptr();

    // 将 CUTLASS 数据复制到 PyTorch Tensor
    hipMemcpy(torch_data_ptr, cutlass_data_ptr, cutlass_data_size, hipMemcpyDeviceToDevice);

    return output;

}
